#include "hip/hip_runtime.h"
#include "common/book.h"


const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin (32, (N + threadsPerBlock-1)/ threadsPerBlock);


// struct for storing matrices
typedef struct {
    int m;
    int n;
    float* elts;
} Matrix;



/*
* kernel to compute dot product between vectors *a and *b. result places in *c.
*/
__device__ void dot( float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while(tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    //set the cache values
    cache[cacheIndex] = temp;
    
    //synchronize threads in this block
    __syncthreads();

    //for reductions, threadsPerBlock must be power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];    
}



/*
* A: matrix (symmetric and positive definite) 
* L: L matrix (lower triangular)
*/
__device__ void cholesky( Matrix A, Matrix L){

} 





/*
* A,B: matrices to multiply
* C: resulting matrix of A*B
*/
__device__ void matrix_multiply( Matrix A, Matrix B, Matrix C){

    //Block row and column
    int row_block = blockIdx.y;
    int col_block = blockIdx.x;

    //each thread block computes submatrix of dimensions BLOCK_SIZE*BLOCK_SIZE
    Matrix sub_c = get_sub_mtx(C, row_block, column_block);

    // each element computes one element of sub matrix sub_c
    // we accumulate the results in val
    float val = 0;

    // thread row and column
    int row = threadIdx.y;
    int col = threadIdx.x;


    // loop the sub matrices of A and B required to compute the sub_c matrix
    // note that this assumes A.n is a multiple of BLOCK_SIZE
    for (int i = 0 ; i < (A.n / BLOCK_SIZE); ++i){
        
        // get sub-mtx sub_a of A and sub_b of B
        Matrix sub_a = get_sub_mtx(A, row_block, i);
        Matrix sub_b = get_sub_mtx(B, i, col_block); 
        
        //TODO complete kernel        
    }
    //TODO finish kerner matrix mult
    
}  


/*
* I_vals: matrix of independent variable values [ [x0,y0,z0...], [x1,y1,z1...], ...]
* b: vector of values corresponsing to elements in I_vals matrix. 
* *r: pointer to result of regression - solution
*/
__global__ void linear_regression( Matrix I_vals, Matrix b, float *r) {
    
}




//TODO linear regression using dot product and inverse/LU/Choelsky




