#include "hip/hip_runtime.h"
#include "common/book.h"
#include <math.h>


#define imin(a,b) (a<b?a:b)
#define imax(a,b) (a>b?a:b)

#define TILE_DIM 3
#define EPSILON 0.000000005

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin (32, (N + threadsPerBlock-1)/ threadsPerBlock);


//const int BLOCK_SIZE = 3;


// struct for storing matrices
typedef struct {
    int m;
    int n;
    int stride;
    float *elts; 
} Matrix;

// struct for vector
typedef struct {
    int length;
    float *elts;
} Vector;




// Some Utility functions

/**
* generates matrix struct with width, height and float elements
*/
Matrix create_matrix( int width, int height, float * elements){
     Matrix A;
     A.n = A.stride = width;
     A.m = height;
     A.elts = elements;
     return A;
}

/**
* generates vector struct with length and elts)
*/
Vector create_vector(int length, float * elements){
    Vector v;
    v.length = length;
    v.elts = elements;
    return v;
}



/**
* Returns true if two matrices are equal
*/
bool mtx_equal(Matrix A, Matrix B){
     if(A.n != B.n || A.m != B.m){
         return false;
     }else{
         int w = A.n;
         int h = A.m;
         for(int z = 0; z < w * h; z++){
             float v1 = A.elts[z];
             float v2 = B.elts[z];
                              
             if((v1 - v2) > EPSILON || (v1-v2) < -EPSILON ){
                  return false;
             }
         }
         return true;
     }
}



/**
* Returns true if two vectors are equal
*/
bool vec_equal(Vector a, Vector b){
     if(b.length != a.length){
          return false;
     }else{
         int len = b.length;
         for(int z = 0; z < len; ++z){
             float v1 = a.elts[z];
             float v2 = b.elts[z];
             
             if((v1-v2) > EPSILON || (v1-v2) < -EPSILON){
                 return false;
             }
         }
         return true;
     }
}




__host__ __device__ void print_matrix( Matrix A){
    printf("--------------------------\n");
    for(int i = 0; i < (A.n * A.m); ++i){
        if( i % A.n == 0) printf("\n");
        printf("%f ", A.elts[i]);
    }
    printf("--------------------------\n");
}


__host__ __device__ void print_vector( Vector v ){
   printf("-------------------------------\n");
   for(int i = 0; i < v.length; ++i){
       printf(" %f \n", v.elts[i]);
   }
   printf("--------------------------------\n");
}



/*
* kernel to compute dot product between vectors *a and *b. result places in *c.
*/
__device__ void dot( float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while(tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    //set the cache values
    cache[cacheIndex] = temp;
    
    //synchronize threads in this block
    __syncthreads();

    //for reductions, threadsPerBlock must be power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];    
}





/*
* S: Matrix to get sub matrix from
* row, column: row and column to start at
* return matrix X which is sub matrix of S
*/
template <int BLOCK_SIZE>
__host__ __device__ Matrix get_sub_mtx( const Matrix S, int row, int col){
    Matrix X;
    X.n = BLOCK_SIZE;
    X.m = BLOCK_SIZE;
    X.stride = S.stride;
    X.elts = &S.elts[S.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return X;
}


// Get a matrix element
__device__ float get_elt(const Matrix A, int row, int col){   
    //printf("returning %f\n ", A.elts[row* A.stride + col]); 
    return A.elts[row * A.stride + col];
}

// Set a matrix element
__device__ void set_elt(Matrix A, int row, int col, float value){
     //printf("setting row*A.stride + col (%d) to value (%f)\n", row*A.stride + col, value);         
     A.elts[row * A.stride + col] = value;
}



/*
* M: matrix of rows of data points (for out use-case this is a column matrix)
* weights: Vector of weights to apply for each row
* R: matrix result of applying weights
*/
__global__ void apply_weights( Matrix M, Vector weights, Matrix R){

    // only use one dimensional block size
    int b_id = blockIdx.x;
    int t_id = threadIdx.x; // each thread takes care of one data point (i.e one row).

    int index = b_id * gridDim.x + t_id;
      
    if(index < (M.m * M.n)) {        
        int row = (index / M.m);
        int col = (index % M.m);
        int val = get_elt( M, row, col);
        int newVal = val * weights.elts[row]; //multiply by the appropriate weight
        set_elt( R, row, col, newVal);  
    }
}







/*
* L: cholesky lower triangle matrix
* U: should be the transpose of L: upper triangular result from cholesky
* b: vector b 
* x: vector x to solve for
*/
__global__ void fwd_bkwd_elimination( Matrix L, Matrix U, Vector b, Vector r){

  
   int n = b.length;
   //__shared__ float y[n]; // representing vector y
   //__shared__ float x[n]; // representing vector x
 
   float *y = (float *)malloc(n * sizeof(float));
   memset(y, 0.0f, n*sizeof(float));
   

    
   //forward solve Ly = b
   for(int i = 0; i < n; ++i){
       y[i] = b.elts[i];
       for(int j = 0; j < i; ++j){
           y[i] -= get_elt( L, i, j) * y[j];
       }
       y[i] /= get_elt(L, i, i);
   }
   
   //backward solve Ux = y
   for(int i = n-1; i > -1; --i){
       r.elts[i] = y[i];
       for(int j = i+1; j < n; j++){
           r.elts[i] -= get_elt(U, i, j) * r.elts[j];
       }
       r.elts[i] /= get_elt( U,i, i);
   }
   

   //printf("printing x\n");
   //for(int i = 0; i< n; i++) { printf("x = %f\n", r.elts[i]);}

   free(y);
}



/*
* A: matrix (symmetric and positive definite) 
* L: L matrix (lower triangular)
*
* This function does not need to be parallelized
* Maybe place this on host
*
* This should pass Matrix L with all 0.0 values
*
*/
__global__ void cholesky( Matrix A, Matrix L){
               
    int n = L.n;
    
    // init matrix
    for(int z = 0; z < n*n ; ++z){
        L.elts[z] = 0.0f;
    } 

    for(int i=0; i < n; ++i){
        for(int k =0; k < i+1; ++k){
            float tmp_sum = 0;
            for(int j = 0; j < k; ++j){
                tmp_sum += ( get_elt( L, i, j) * get_elt(L, k, j));
            }
            if(i == k){
                float v = sqrt( get_elt(A, i, i) - tmp_sum );
                //printf("v = %f\n", v);                
                set_elt(L, i, k, v);
            }else{
		float v = 1.0 / get_elt(L, k , k) * (get_elt(A, i, k) - tmp_sum);
	        //printf("v = %f \n", v);
                set_elt(L, i , k, v);
            }  
        }
    }      
}




/*
* Compute matrix transpose
* This relies on the small side of matrix being less than maximum grid dimension
*/
template <int BLOCK_SIZE>
__global__ void matrix_transpose(Matrix A, Matrix At){
   
   int row_block = blockIdx.x;
   int col_block = blockIdx.y;   
  

   int row, _row, col, index;
   
   if ( A.m >= A.n ){
       row = (row_block * BLOCK_SIZE + threadIdx.x);
       _row = row * A.n;
       col = col_block;
       index = _row + col;
       if (row < A.m){
           float elt = A.elts[index];
           At.elts[ col * A.m + row] = elt;
       }            

   } else {
       row = row_block;
       _row = row * A.n;
       col = col_block * BLOCK_SIZE + threadIdx.x;
       index = _row + col;//row_block * A.n + col_block * BLOCK_SIZE + threadIdx.x;
       if ( col < A.n) {
            float elt = A.elts[index];
            At.elts[ col * A.m + row ] = elt;
       }
   }

}



/*
* A: matrix to transpose
* At: tranposed Matrix
*/
template <int BLOCK_SIZE>
__global__ void matrix_transpose_x(Matrix A, Matrix At){
  
   //Block row and column
   int row_block = blockIdx.y;
   int col_block = blockIdx.x;
  
   //diagonal reorder for transpose implementation
   int block_x, block_y;
   
   if(A.n == A.m){
       block_y = blockIdx.x;
       block_x = (blockIdx.y + blockIdx.x) % gridDim.x;
   } else {
       block_y = ((blockIdx.y * gridDim.x) + blockIdx.x) % gridDim.y;
       block_x = ((((blockIdx.y * gridDim.x) + blockIdx.x)/gridDim.y) + block_y) % gridDim.x;
   }
   
   int x = block_x * BLOCK_SIZE + threadIdx.x;
   int y = block_y * BLOCK_SIZE + threadIdx.y;

   int in_index = x + y * A.n; 
   
   x = block_y * BLOCK_SIZE + threadIdx.x;
   y = block_x * BLOCK_SIZE + threadIdx.y;

   int out_index = x + y * A.m;
   
   // sub matrix of result transpose
   //Matrix sub_at = get_sub_mtx(At, blockIdx.y, blockIdx.x);
   
   // thread row and column
   int row = threadIdx.y;
   int col = threadIdx.x;


   
   
   int nVal = (A.n > A.m)? A.n : A.m;
   

   for(int j = 0; j < ceil(nVal / (float)BLOCK_SIZE); ++j){
   
       __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE+1];
       
       Matrix sub_a, sub_at;
       //sub_a = get_sub_mtx(A, block_y, block_x);
       //sub_at = get_sub_mtx(At, block_y, block_x);
           
              
       if( A.n >= A.m){
           sub_a = get_sub_mtx<BLOCK_SIZE>(A, block_y, j);               
           sub_at = get_sub_mtx<BLOCK_SIZE>(At, j , block_x);
       } else {
           sub_a = get_sub_mtx<BLOCK_SIZE>(A, j, block_x);
           sub_at = get_sub_mtx<BLOCK_SIZE>(At, block_y , j);
       } 
      
       /*      
       printf("this is iteration %f\n", j);
       printf("sub matrix a\n");
       print_matrix(sub_a);
       printf("sub matrix at\n");
       print_matrix(sub_at);*/
         
         
       for (int i = 0; i < BLOCK_SIZE; i+= BLOCK_SIZE){
           //tile[threadIdx.y + i][threadIdx.x] = A.elts[in_index + i*A.n];
           tile[row + i][col] = sub_a.elts[in_index + i*A.n];//get_elt(sub_a, row, in_index);
           
       }   
   
       __syncthreads();
  
       for(int i = 0; i < BLOCK_SIZE; i+=BLOCK_SIZE){
           sub_at.elts[out_index + i * A.m] = tile[col][row + i];
           //__syncthreads();
       }
       
       __syncthreads();      
         
   }
   //__syncthreads();

}


/*
* A: Matrix A
* x: vector x to multiply matrix A by
* b: result of multiplying A*x.  b = A*x
*/
//__global__ void matrix_multiply_vector(Matrix A, Vector x, Vector b){
   


//}



/*
* A,B: matrices to multiply
* C: resulting matrix of A*B
*/
template <int BLOCK_SIZE>
__global__ void matrix_multiply_matrix( Matrix A, Matrix B, Matrix C){

    //Block row and column
    int row_block = blockIdx.y;
    int col_block = blockIdx.x;

    //each thread block computes submatrix of dimensions BLOCK_SIZE*BLOCK_SIZE
    Matrix sub_c = get_sub_mtx<BLOCK_SIZE>(C, row_block, col_block);
     
    
    // each thread computes one element of sub matrix sub_c
    // we accumulate the results in val
    float val = 0;

    // thread row and column
    int row = threadIdx.y;
    int col = threadIdx.x;

    
    
    // loop the sub matrices of A and B required to compute the sub_c matrix
    // note that this assumes A.n is a multiple of BLOCK_SIZE
    for (int i = 0 ; i < ceil(A.n / (float)BLOCK_SIZE); ++i){
        
        // get sub-mtx sub_a of A and sub_b of B
        Matrix sub_a = get_sub_mtx<BLOCK_SIZE>(A, row_block, i);
        Matrix sub_b = get_sub_mtx<BLOCK_SIZE>(B, i, col_block); 
                
        
        //shared memory to fill sub matrices 
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];       
       
        
        As[row][col] = get_elt(sub_a, row, col);
        Bs[row][col] = get_elt(sub_b, row, col);
        
        //printf("sub_a elt at row %d and col %d = %f \n ", row, col, get_elt(sub_a, row, col));
        //printf("sub_b elt at row %d and col %d = %f \n ", row, col, get_elt(sub_b, row, col));
        
 
	//synchronize the threads
  	__syncthreads();

        if( (i ==  ceil(A.n/ (float)BLOCK_SIZE) - 1) && A.n % BLOCK_SIZE != 0){
            for(int e = 0; e < A.n % BLOCK_SIZE; ++e){
                val += As[row][e] * Bs[e][col];
            }
        } else{        
            // multiply sub matrices
            for(int e = 0; e < BLOCK_SIZE; ++e){
                val += As[row][e] * Bs[e][col];           
	    }
        }    
        
        
        // synchronize threads 
        __syncthreads();

               
    }
    
    
    // set sub-matrix c_sub element at row, column to val
    // each thread does the following
    set_elt( sub_c, row, col, val);

     
}  


/*
* A: matrix of independent variable values [ [1,x0,y0,z0...], [1, x1,y1,z1...], ...] now it assumes you input the pre- ones (for computing constants)
* b: vector of values corresponsing to elements in A matrix
* return a vector representing result equation    C + Dx + Ey ...
*/
template <int BLOCK_SIZE>
__host__ Vector linear_regression( Matrix A, Matrix b) {
   
    // The following code is to transpose the matrix A
    // Invokes the matrix_transpose kernel
    
    int width_A = A.n;
    int height_A = A.m;
    size_t size_A = width_A * height_A * sizeof(float);
    float * elements_A = A.elts;
         
        
    bool w_gt = width_A > height_A;
    int grid_x = (w_gt)? height_A : ceil(height_A / (float)BLOCK_SIZE);
    int grid_y = (w_gt)? ceil(width_A / (float)BLOCK_SIZE) : width_A;
 
    dim3 dimGrid(grid_x, grid_y);//dim3 dimGrid( height_A, ceil(width_A / (float)BLOCK_SIZE));
    dim3 dimBlock( BLOCK_SIZE);


    Matrix d_A = create_matrix(width_A, height_A, elements_A);
    hipMalloc(&d_A.elts, size_A);
    hipMemcpy(d_A.elts, A.elts, size_A, hipMemcpyHostToDevice); 
    
    float * elts_r = (float *)malloc(size_A);
    memset( elts_r, 0.0f, size_A);
    Matrix d_R = create_matrix(height_A, width_A, elts_r);
    hipMalloc(&d_R.elts, size_A);

    //invoke tranpose kernel
    matrix_transpose<BLOCK_SIZE><<<dimGrid, dimBlock>>>(d_A, d_R);
    
    
    float * elts_at = (float *)malloc(size_A);
    Matrix At = create_matrix(height_A, width_A, elts_at);
    hipMemcpy(At.elts, d_R.elts, size_A, hipMemcpyDeviceToHost);

    printf("matrix A is \n");
    print_matrix(A);
    printf("matrix At is \n");
    print_matrix(At);
          
   
    //free(elts_c);
    free(elts_r);
    hipFree(d_A.elts);
    hipFree(d_R.elts);
    
    // Now we need to multiple the matrices At * A (non-weigheted regression)
    // invoke the matrix_multiply_matrix kernel
    int width_At = At.n;
    int height_At = At.m;
    size_t size_At = size_A;
    float * elements_At = At.elts;

    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2( ceil(height_At / (float)dimBlock2.x), ceil(width_A/ (float)dimBlock2.y));

    Matrix d_At = create_matrix(width_At, height_At, elements_At);
    hipMalloc(&d_At.elts, size_At);
    hipMemcpy(d_At.elts, At.elts, size_At, hipMemcpyHostToDevice);

    hipMalloc(&d_A.elts, size_A);
    hipMemcpy(d_A.elts, A.elts, size_A, hipMemcpyHostToDevice);

    size_t size_C = width_A * height_At * sizeof(float);
    float * elements_C = (float *)malloc(size_C);
    memset(elements_C, 0.0f, size_C);
    Matrix d_R2 = create_matrix(width_A, height_At, elements_C);
    hipMalloc(&d_R2.elts, size_C);
    
    // invoke matrix_multiply_matrix kernel
    matrix_multiply_matrix<BLOCK_SIZE><<<dimGrid2, dimBlock2>>>(d_At, d_A, d_R2);
    
    float * elts_c = (float *)malloc(size_C);
    Matrix C = create_matrix(width_A, height_At, elts_c);
    hipMemcpy(C.elts, d_R2.elts, size_C, hipMemcpyDeviceToHost);
    
    printf("matrix At is  \n");
    print_matrix(At);
    printf("matrix A is   \n");
    print_matrix(A);
    printf("matrix C = At * A is \n");
    print_matrix(C);
    
    
    free(elements_C);
    hipFree(d_R2.elts);
    hipFree(d_A.elts);
    hipFree(d_At.elts);

    

    // Next, multiply the matrix At (tranpose of A) by the vector b. (non-weighted)
    // This also uses the matrix multiply kernel
    
    int width_b = b.n; // should be 1 (b is a vector)
    int height_b = b.m;
    size_t size_b = width_b * height_b * sizeof(float);
    float * elements_b = b.elts;
 
    dim3 dimBlock3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid3(ceil(height_At / (float)dimBlock3.x), ceil(width_b / (float)dimBlock3.y));


    hipMalloc(&d_At.elts, size_At);
    hipMemcpy(d_At.elts, At.elts, size_At, hipMemcpyHostToDevice);

    Matrix d_b = create_matrix(width_b, height_b, elements_b);
    hipMalloc(&d_b.elts, size_b);
    hipMemcpy(d_b.elts, b.elts, size_b, hipMemcpyHostToDevice);

    size_t size_At_b = width_b * height_At * sizeof(float);
    float * elements_At_b = (float *)malloc(size_At_b);
    memset(elements_At_b, 0.0f, size_At_b);
    Matrix d_At_b = create_matrix(width_b, height_At, elements_At_b);
    hipMalloc(&d_At_b.elts, size_At_b);      

    //invoke matrix_multiply_matrix kernel (multiply by vector instead)
    matrix_multiply_matrix<BLOCK_SIZE><<<dimGrid3, dimBlock3>>>(d_At, d_b, d_At_b);

    float * elts_at_b = (float *)malloc(size_At_b);
    Vector vb = create_vector( width_b * height_At, elts_at_b);
    hipMemcpy( vb.elts, d_At_b.elts, size_At_b, hipMemcpyDeviceToHost);

    printf("printing matrix At\n");
    print_matrix(At);
    printf("printing matrix/vector b\n");
    print_matrix(b);
    printf("vector result vb\n");
    print_vector(vb);


    
    free(elements_At_b);
    hipFree(d_At.elts);
    hipFree(d_b.elts);
    hipFree(d_At_b.elts);
    

    // Compute Cholesky for At*A result
    // invoke the cholesky kernel
    int width_C = C.n;
    int height_C = C.m;
    
    Matrix d_C = create_matrix(width_C, height_C, C.elts);
    hipMalloc(&d_C.elts, size_C);
    hipMemcpy(d_C.elts, C.elts, size_C, hipMemcpyHostToDevice);   

    elts_r = (float *)malloc(size_C);
    memset(elts_r, 0.0f, size_C);
    Matrix d_L = create_matrix(width_C, height_C, elts_r);
    hipMalloc(&d_L.elts, size_C);

    //invoke the kernel here
    cholesky<<<1, 1>>>(d_C, d_L);
    
    float * elts_l =  (float *)malloc(size_C);
    Matrix L = create_matrix(width_C, height_C, elts_l);
    hipMemcpy(L.elts, d_L.elts, size_C, hipMemcpyDeviceToHost);

    printf("matrix C is \n");
    print_matrix(C);
    printf("cholesky result L \n");
    print_matrix(L);

    free(elts_r);
    hipFree(d_C.elts);
    hipFree(d_L.elts);

    // compute Lt which is transpose of L matrix (cholesky result)
    // invoke the matrix_transpose kernel 
    int width_L = L.n;
    int height_L = L.m;
    size_t size_L = width_L * height_L * sizeof(float);
    float * elements_L = L.elts;

    grid_x = ceil(height_L / (float)BLOCK_SIZE);
    grid_y = width_L;
    dim3 dimGrid4(grid_x, grid_y);
    dim3 dimBlock4(BLOCK_SIZE); 
   
    hipMalloc(&d_L.elts, size_L);
    hipMemcpy(d_L.elts, L.elts, size_L, hipMemcpyHostToDevice);

    elts_r = (float *)malloc(size_L);
    memset(elts_r, 0.0f, size_L);
    Matrix d_U = create_matrix(height_L, width_L, elts_r);
    hipMalloc(&d_U.elts, size_L);

    //invoke kernel
    matrix_transpose<BLOCK_SIZE><<<dimGrid4, dimBlock4>>>(d_L, d_U);
   
    float * elts_u = (float *)malloc(size_L);
    Matrix U = create_matrix(height_L, width_L, elts_u);
    hipMemcpy(U.elts, d_U.elts, size_L, hipMemcpyDeviceToHost);

    printf("matrix L is \n");
    print_matrix(L);
    printf("matrix U is \n");
    print_matrix(U);

    free(elts_r);
    hipFree(d_L.elts);
    hipFree(d_U.elts);

    // compute result Vector x by forward-backward elimination
    // invoke forward-backward elimination kernel
    
    int width_U = U.n;
    int height_U = U.m;
    size_t size_U = width_U * height_U * sizeof(float);
    float * elements_U = U.elts;
     
    int v_len = vb.length;
    float * v_elts = vb.elts;

    size_t size_v = v_len * sizeof(float);
    
    hipMalloc(&d_L.elts, size_L);
    hipMemcpy(d_L.elts, L.elts, size_L, hipMemcpyHostToDevice);

    hipMalloc(&d_U.elts, size_U);
    hipMemcpy(d_U.elts, U.elts, size_U, hipMemcpyHostToDevice);

    Vector d_vb = create_vector(v_len, v_elts);
    hipMalloc(&d_vb.elts, size_v);
    hipMemcpy(d_vb.elts, vb.elts, size_v, hipMemcpyHostToDevice);

    elts_r = (float *)malloc(size_v);
    memset(elts_r, 0.0f, size_v);
    Vector d_r = create_vector(v_len, elts_r);
    hipMalloc(&d_r.elts, size_v);


    //invoke kernel
    fwd_bkwd_elimination<<<1, 1>>>(d_L, d_U, d_vb, d_r);

    float * elts_x = (float *)malloc(size_v);
    memset(elts_x, 0.0f, size_v);
    Vector x = create_vector(v_len, elts_x);
    hipMemcpy(x.elts, d_r.elts, size_v, hipMemcpyDeviceToHost);


    printf("matrix L\n");
    print_matrix(L);
    printf("matrix U\n");
    print_matrix(U);
    printf("vector vb\n");
    print_vector(vb);
    printf("vector x result \n");
    print_vector(x);

    //Vector result = create_vector(x.length, x.elts);

    free(elts_c);    
    free(elts_r);
    free(elts_l);
    free(elts_u);
    free(elts_at_b);    
    hipFree(d_vb.elts);
    hipFree(d_L.elts);
    hipFree(d_U.elts);
    hipFree(d_r.elts);

    return x;
}






//int main ( void ) {
    
    /*   
    // test matrix multiplication (A_T * A)
            
    printf("matrix multiply test\n");
    //Square matrix test    
    int width = 3;
    int height = 3;
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(height / dimBlock.x, height / dimBlock.y);
            
    Matrix A;
    A.n = width;
    A.m = height;
    float a[9] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};
    A.elts = a;
           

    Matrix At;
    At.n = height;
    At.m = width;
    float at[9] = {1.0f, 4.0f, 7.0f, 2.0f, 5.0f, 8.0f, 3.0f, 6.0f, 9.0f};
    At.elts = at;
       

    Matrix d_A;
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size_t size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    Matrix d_At;
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f};
    d_C.elts = c;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid, dimBlock>>>(d_At, d_A, d_C);

    
    Matrix C;
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
  
        
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 
    

    //column matrix A test:  A_T * A is row matrix * column matrix 
  
    width = 3;
    height = 9;
    
    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2(width / dimBlock.x, width / dimBlock.y);
            
    
    A.n = width;
    A.m = height;
    float a2[27] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f,1.0f, 2.0f, 3.0f, 4.0f, 5.0f,6.0f, 7.0f, 8.0f, 9.0f };
    A.elts = a2;
           

    
    At.n = height;
    At.m = width;
    float at2[27] = {1.0f, 4.0f, 7.0f, 1.0f, 4.0f, 7.0f, 1.0f, 4.0f, 7.0f, 2.0f, 5.0f, 8.0f, 2.0f, 5.0f, 8.0f, 2.0f, 5.0f, 8.0f,3.0f, 6.0f, 9.0f, 3.0f, 6.0f,9.0f, 3.0f, 6.0f, 9.0f };
    At.elts = at2;
       

    
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c2[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f };
    d_C.elts = c2;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid2, dimBlock2>>>(d_At, d_A, d_C);// A_t * A

        
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
      
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 


    // test A matrix row matrix -- A_t is column mtx 
   
  
    width = 6;
    height = 3;
    
    dim3 dimBlock3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid3(width / dimBlock.x, width / dimBlock.y);
            
    
    A.n = width;
    A.m = height;
    float a3[18] = {1.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 3.0f, 3.0f, 3.0f, 1.0f, 1.0f, 1.0f, 4.0f, 4.0f, 4.0f, 1.0f, 1.0f, 1.0f };
    A.elts = a3;
           

    
    At.n = height;
    At.m = width;
    float at3[18] = {1.0f, 3.0f, 4.0f, 1.0f, 3.0f, 4.0f, 1.0f, 3.0f, 4.0f, 2.0f, 1.0f, 1.0f, 2.0f, 1.0f, 1.0f, 2.0f, 1.0f, 1.0f };
    At.elts = at3;
       

    
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c3[36] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
    d_C.elts = c3;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid3, dimBlock3>>>(d_At, d_A, d_C);// A_t * A

        
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
      
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 
       
    // test matrix multiply vector 

    
    */    
    /*    
             
    // Test matrix transpose
      
    printf("matrix transpose test\n");


    Matrix X;
    X.n = 3;
    X.m = 6;
    float a[18] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f};
    X.elts = a;
    
    Matrix d_X;
    d_X.n = d_X.stride = X.n;
    d_X.m = X.m;
    size_t size = X.n * X.m * sizeof(float);
    
    hipMalloc(&d_X.elts, size);
    hipMemcpy(d_X.elts, X.elts, size, hipMemcpyHostToDevice);

    
    Matrix d_Xt;
    d_Xt.n = d_Xt.stride =  X.m;
    d_Xt.m = X.n; 
    size = d_Xt.m * d_Xt.n * sizeof(float);
    float c[18] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    d_Xt.elts = c;
    hipMalloc(&d_Xt.elts, size);

    
    // Invoke Kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(X.n / BLOCK_SIZE, X.m / BLOCK_SIZE);

    matrix_transpose<<<dimGrid, dimBlock>>>(d_X, d_Xt);

    Matrix Xt;
    Xt.m = d_Xt.m;
    Xt.n = Xt.stride = d_Xt.n;
    Xt.elts = (float *)malloc(sizeof(float) * Xt.m * Xt.n);
    hipMemcpy(Xt.elts, d_Xt.elts, size, hipMemcpyDeviceToHost);
    
    //print_matrix(X);
    //print_matrix(X);
    print_matrix(X);
    printf("transpose result\n");
    print_matrix(Xt);    

    free(Xt.elts);
    hipFree(d_X.elts);
    hipFree(d_Xt.elts);
    
    */
    /*   
    // test matrix multiplication (A_T * A)
    
    printf("cholesky decomp tets matrix multiply test\n");
    //Square matrix test    
    int width = 3;
    int height = 3;
    
    //dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    //dim3 dimGrid(height / dimBlock.x, height / dimBlock.y);
            
    Matrix C;
    C.n = C.stride = width;
    C.m = height;
    float c[9] = {4.0f, 12.0f, -16.0f, 12.0f, 37.0f, -43.0f, -16.0f, -43.0f, 98.0f};
    //float c[9] = {25.0f, 15.0f, -5.0f, 15.0f, 18.0f, 0.0f, -5.0f, 0.0f, 11.0f};
    C.elts = c;
   
    Matrix d_C;
    d_C.n = d_C.stride = C.n;
    d_C.m = C.m;
    size_t size_C = C.n * C.m * sizeof(float);
    hipMalloc(&d_C.elts, size_C);
    hipMemcpy(d_C.elts, C.elts, size_C, hipMemcpyHostToDevice);

    Matrix d_L;
    d_L.n = d_L.stride = C.n;
    d_L.m = C.m;
    size_t size_L = d_C.m * d_C.n * sizeof(float);
    float dL[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f};
    d_L.elts = dL;
    hipMalloc(&d_L.elts, size_L);

    

    cholesky<<<1,1>>>(d_C, d_L);


    Matrix L;
    L.n = L.stride = C.n;
    L.m = C.m;
    float _l[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    L.elts = _l;
    //L.elts = (float *)malloc( sizeof(float) * C.m * C.n);
    hipMemcpy(L.elts, d_L.elts, size_L, hipMemcpyDeviceToHost);

    printf("start cholesky for C matrix: \n");
    print_matrix(C);
    printf("cholesky result\n");    
    print_matrix(L);


    //free(L.elts);
    hipFree(d_L.elts);
    hipFree(d_C.elts);
   */
    
 
//    return 0;
//}


