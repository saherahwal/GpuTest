#include "hip/hip_runtime.h"
#include "common/book.h"

#define imin(a,b) (a<b?a:b)
#define BLOCK_SIZE 3
#define TILE_DIM 3

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin (32, (N + threadsPerBlock-1)/ threadsPerBlock);


// struct for storing matrices
typedef struct {
    int m;
    int n;
    int stride;
    float *elts; 
} Matrix;


__host__ __device__ void print_matrix( Matrix A){
    printf("--------------------------\n");
    for(int i = 0; i < (A.n * A.m); ++i){
        if( i % A.n == 0) printf("\n");
        printf("%f ", A.elts[i]);
    }
    printf("--------------------------\n");
}



/*
* kernel to compute dot product between vectors *a and *b. result places in *c.
*/
__device__ void dot( float *a, float *b, float *c){
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while(tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    //set the cache values
    cache[cacheIndex] = temp;
    
    //synchronize threads in this block
    __syncthreads();

    //for reductions, threadsPerBlock must be power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];    
}



/*
* A: matrix (symmetric and positive definite) 
* L: L matrix (lower triangular)
*/
__device__ void cholesky( Matrix A, Matrix L){

} 


/*
* S: Matrix to get sub matrix from
* row, column: row and column to start at
* return matrix X which is sub matrix of S
*/
__host__ __device__ Matrix get_sub_mtx( const Matrix S, int row, int col){
    Matrix X;
    X.n = BLOCK_SIZE;
    X.m = BLOCK_SIZE;
    X.stride = S.stride;
    X.elts = &S.elts[S.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
    return X;
}


// Get a matrix element
__device__ float get_elt(const Matrix A, int row, int col){   
    //printf("returning %f\n ", A.elts[row* A.stride + col]); 
    return A.elts[row * A.stride + col];
}

// Set a matrix element
__device__ void set_elt(Matrix A, int row, int col, float value){         
     A.elts[row * A.stride + col] = value;
}


/*
* A: matrix to transpose
* At: tranposed Matrix
*/
__global__ void matrix_transpose(Matrix A, Matrix At){
  
     
   __shared__ float tile[TILE_DIM][TILE_DIM+1];
   
   //diagonal reorder for transpose implementation
     
   int block_x, block_y;
   
   if(A.n == A.m){
       block_y = blockIdx.x;
       block_x = (blockIdx.y + blockIdx.x) % gridDim.x;
   } else {
       block_y = ((blockIdx.y * gridDim.x) + blockIdx.x) % gridDim.y;
       block_x = ((((blockIdx.y * gridDim.x) + blockIdx.x)/gridDim.y) + block_y) % gridDim.x;
   }
   
   int x = block_x * TILE_DIM + threadIdx.x;
   int y = block_y * TILE_DIM + threadIdx.y;

   int in_index = x + y * A.n; 
   
   x = block_y * TILE_DIM + threadIdx.x;
   y = block_x * TILE_DIM + threadIdx.y;

   int out_index = x + y * A.m;
  
 
   


   for (int i = 0; i < TILE_DIM; i+= BLOCK_SIZE){
       tile[threadIdx.y + i][threadIdx.x] = A.elts[in_index + i*A.n];
   }
   
   
   __syncthreads();
  
   for(int i = 0; i < TILE_DIM; i+= BLOCK_SIZE){
       printf("out_index = %d \n", out_index);
       printf("i = %d \n", i);
       printf("threadIdx.x = %d \n", threadIdx.x);
       printf("threadIdx.y = %d \n", threadIdx.y);
       printf("tile[tidx][tidy + i]= %f \n", tile[threadIdx.x][threadIdx.y + i]);
       At.elts[out_index + i * A.m] = tile[threadIdx.x][threadIdx.y + i];
       printf("I am here bro!");  
       
   }
   
   //printf("printing matrix from cuda");
   //print_matrix(At);   
   
}











/*
* A,B: matrices to multiply
* C: resulting matrix of A*B
*/
__global__ void matrix_multiply_matrix( Matrix A, Matrix B, Matrix C){

    //Block row and column
    int row_block = blockIdx.y;
    int col_block = blockIdx.x;

    //each thread block computes submatrix of dimensions BLOCK_SIZE*BLOCK_SIZE
    Matrix sub_c = get_sub_mtx(C, row_block, col_block);
     
    
    // each element computes one element of sub matrix sub_c
    // we accumulate the results in val
    float val = 0;

    // thread row and column
    int row = threadIdx.y;
    int col = threadIdx.x;

    
    
    // loop the sub matrices of A and B required to compute the sub_c matrix
    // note that this assumes A.n is a multiple of BLOCK_SIZE
    for (int i = 0 ; i < (A.n / BLOCK_SIZE); ++i){
        
        // get sub-mtx sub_a of A and sub_b of B
        Matrix sub_a = get_sub_mtx(A, row_block, i);
        Matrix sub_b = get_sub_mtx(B, i, col_block); 
         
        
        
        //shared memory to fill sub matrices 
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];       
        
        As[row][col] = get_elt(sub_a, row, col);
        Bs[row][col] = get_elt(sub_b, row, col);
        
        //printf("sub_a elt at row %d and col %d = %f \n ", row, col, get_elt(sub_a, row, col));
        //printf("sub_b elt at row %d and col %d = %f \n ", row, col, get_elt(sub_b, row, col));
        
 
	//synchronize the threads
  	__syncthreads();

                
        // multiply sub matrices
        for(int e = 0; e < BLOCK_SIZE; ++e){
            val += As[row][e] * Bs[e][col];
            
	}    
        
        
        // synchronize threads 
        __syncthreads();

               
    }
    
    
    // set sub-matrix c_sub element at row, column to val
    // each thread does the following
    set_elt( sub_c, row, col, val);

     
}  


/*
* I_vals: matrix of independent variable values [ [x0,y0,z0...], [x1,y1,z1...], ...]
* b: vector of values corresponsing to elements in I_vals matrix. 
* *r: pointer to result of regression - solution
*/
__global__ void linear_regression( Matrix I_vals, Matrix b, float *r) {
    
}



//TODO linear regression using dot product and inverse/LU/Choelsky












int main ( void ) {
    
    
    // test matrix multiplication (A_T * A)
    
    printf("matrix multiply test\n");
    //Square matrix test    
    int width = 3;
    int height = 3;
    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(height / dimBlock.x, height / dimBlock.y);
            
    Matrix A;
    A.n = width;
    A.m = height;
    float a[9] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f};
    A.elts = a;
           

    Matrix At;
    At.n = height;
    At.m = width;
    float at[9] = {1.0f, 4.0f, 7.0f, 2.0f, 5.0f, 8.0f, 3.0f, 6.0f, 9.0f};
    At.elts = at;
       

    Matrix d_A;
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size_t size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    Matrix d_At;
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f};
    d_C.elts = c;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid, dimBlock>>>(d_At, d_A, d_C);

    
    Matrix C;
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
  
        
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 
    

    //column matrix A test:  A_T * A is row matrix * column matrix 
  
    width = 3;
    height = 9;
    
    dim3 dimBlock2(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid2(width / dimBlock.x, width / dimBlock.y);
            
    
    A.n = width;
    A.m = height;
    float a2[27] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f,1.0f, 2.0f, 3.0f, 4.0f, 5.0f,6.0f, 7.0f, 8.0f, 9.0f };
    A.elts = a2;
           

    
    At.n = height;
    At.m = width;
    float at2[27] = {1.0f, 4.0f, 7.0f, 1.0f, 4.0f, 7.0f, 1.0f, 4.0f, 7.0f, 2.0f, 5.0f, 8.0f, 2.0f, 5.0f, 8.0f, 2.0f, 5.0f, 8.0f,3.0f, 6.0f, 9.0f, 3.0f, 6.0f,9.0f, 3.0f, 6.0f, 9.0f };
    At.elts = at2;
       

    
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c2[9] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f };
    d_C.elts = c2;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid2, dimBlock2>>>(d_At, d_A, d_C);// A_t * A

        
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
      
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 


    // test A matrix row matrix -- A_t is column mtx 
   
  
    width = 6;
    height = 3;
    
    dim3 dimBlock3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid3(width / dimBlock.x, width / dimBlock.y);
            
    
    A.n = width;
    A.m = height;
    float a3[18] = {1.0f, 1.0f, 1.0f, 2.0f, 2.0f, 2.0f, 3.0f, 3.0f, 3.0f, 1.0f, 1.0f, 1.0f, 4.0f, 4.0f, 4.0f, 1.0f, 1.0f, 1.0f };
    A.elts = a3;
           

    
    At.n = height;
    At.m = width;
    float at3[18] = {1.0f, 3.0f, 4.0f, 1.0f, 3.0f, 4.0f, 1.0f, 3.0f, 4.0f, 2.0f, 1.0f, 1.0f, 2.0f, 1.0f, 1.0f, 2.0f, 1.0f, 1.0f };
    At.elts = at3;
       

    
    d_A.n = d_A.stride = A.n;
    d_A.m = A.m;
    size = A.n * A.m * sizeof(float);
    
    hipMalloc(&d_A.elts, size);
    hipMemcpy(d_A.elts, A.elts, size, hipMemcpyHostToDevice);
        
    
    d_At.n = d_At.stride = At.n;
    d_At.m = At.m;
    
    hipMalloc(&d_At.elts, size);
    hipMemcpy(d_At.elts, At.elts, size, hipMemcpyHostToDevice);
    
    
    d_C.n = d_C.stride =  A.n;
    d_C.m = A.n; // square matrix
    size = d_C.m * d_C.n * sizeof(float);
    float c3[36] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
    d_C.elts = c3;
    hipMalloc(&d_C.elts, size);

    //invoke kernel
    matrix_multiply_matrix<<<dimGrid3, dimBlock3>>>(d_At, d_A, d_C);// A_t * A

        
    C.m = At.m;
    C.n = A.n;
    C.elts = (float *)malloc(sizeof(float) * C.m * C.n);
    hipMemcpy(C.elts, d_C.elts, size, hipMemcpyDeviceToHost);
   
        
    print_matrix(A);
    print_matrix(At);
    print_matrix(C);    
      
     
    free(C.elts);    
    hipFree(d_A.elts);
    hipFree(d_At.elts);
    hipFree(d_C.elts); 
        
        
             
    // Test matrix transpose
    
    /*


    Matrix X;
    X.n = 6;
    X.m = 3;
    float a[18] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f, 6.0f, 7.0f, 8.0f, 9.0f, 10.0f, 11.0f, 12.0f, 13.0f, 14.0f, 15.0f, 16.0f, 17.0f, 18.0f};
    X.elts = a;
    
    Matrix d_X;
    d_X.n = d_X.stride = X.n;
    d_X.m = X.m;
    size_t size = X.n * X.m * sizeof(float);
    
    hipMalloc(&d_X.elts, size);
    hipMemcpy(d_X.elts, X.elts, size, hipMemcpyHostToDevice);

    
    Matrix d_Xt;
    d_Xt.n = d_Xt.stride =  X.m;
    d_Xt.m = X.n; 
    size = d_Xt.m * d_Xt.n * sizeof(float);
    float c[18] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,0.0f,0.0f, 0.0f,0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    d_Xt.elts = c;
    hipMalloc(&d_Xt.elts, size);

    
    // Invoke Kernel
    dim3 dimBlock(TILE_DIM, BLOCK_SIZE);
    dim3 dimGrid(X.m / TILE_DIM, X.n / TILE_DIM);

    matrix_transpose<<<dimGrid, dimBlock>>>(d_X, d_Xt);

    Matrix Xt;
    Xt.m = d_Xt.m;
    Xt.n = Xt.stride = d_Xt.n;
    Xt.elts = (float *)malloc(sizeof(float) * Xt.m * Xt.n);
    hipMemcpy(Xt.elts, d_Xt.elts, size, hipMemcpyDeviceToHost);

    printf("transpose result");
    print_matrix(Xt);    

    free(Xt.elts);
    hipFree(d_X.elts);
    hipFree(d_Xt.elts);

    */
 
    return 0;
}





